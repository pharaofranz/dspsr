#include <vector>

#include "catch.hpp"

#include "dsp/InverseFilterbank.h"
#include "dsp/InverseFilterbankEngine.h"
#include "dsp/InverseFilterbankEngineCPU.h"
#include "dsp/InverseFilterbankEngineCUDA.h"
#include "dsp/MemoryCUDA.h"

#include "util.hpp"
#include "InverseFilterbankTestConfig.hpp"

static util::InverseFilterbank::InverseFilterbankTestConfig test_config;

void check_error (const char*);

class Reporter : public dsp::InverseFilterbank::Engine::Reporter {
public:

  Reporter (bool _iscuda = false): iscuda(_iscuda) {}

  Reporter (hipStream_t _stream): stream(_stream) { iscuda = true; }

  void operator() (float* arr, unsigned nchan, unsigned npol, unsigned ndat, unsigned ndim)
  {
    unsigned total_size = nchan * npol * ndat * ndim;
    if (util::config::verbose)
    {
      std::cerr << "Reporter::operator() ("
        << arr << ", "
        << nchan << ", "
        << npol << ", "
        << ndat << ", "
        << ndim << ")"
        << std::endl;
      std::cerr << "Reporter::operator() total_size=" << total_size << std::endl;
    }
    std::vector<float> data (total_size);
    if (iscuda) {
      float* data_ptr = data.data();
      size_t total_size_bytes = total_size * sizeof(float);
      hipError_t error;
      if (stream) {
        error = hipMemcpyAsync(data_ptr, arr, total_size_bytes, hipMemcpyDeviceToHost, stream);
        if (error != 0) {
          throw "hipMemcpyAsync error";
        }
        error = hipStreamSynchronize(stream);
      } else {
        error = hipMemcpy((void*) data_ptr, (void*) arr, total_size_bytes, hipMemcpyDeviceToHost);
        if (error != 0) {
          throw "hipMemcpy error";
        }
        error = hipDeviceSynchronize();
      }
      check_error("Reporter::operator()");
    } else {
      if (util::config::verbose) {
        std::cerr << "Reporter::operator() assigning vector contents" << std::endl;
      }
      data.assign(arr, arr + total_size);
    }
    data_vectors.push_back(data);
  }

  hipStream_t stream;
  bool iscuda;
  std::vector<std::vector<float>> data_vectors;

};



TEST_CASE (
  "InverseFilterbankEngineCPU and InverseFilterbankEngineCUDA produce same output",
  "[InverseFilterbankEngineCPU]"
)
{

  std::vector<float> thresh = test_config.get_thresh();
  std::vector<util::TestShape> test_shapes = test_config.get_test_vector_shapes();
  auto idx = GENERATE_COPY(range(0, (int) test_shapes.size()));
  util::TestShape test_shape = test_shapes[idx];


  void* stream = 0;
  hipStream_t hip_stream = reinterpret_cast<hipStream_t>(stream);
  CUDA::DeviceMemory* device_memory = new CUDA::DeviceMemory(hip_stream);
  CUDA::InverseFilterbankEngineCUDA engine_cuda(hip_stream);
  dsp::InverseFilterbankEngineCPU engine_cpu;

  Reporter reporter_cpu;
  Reporter reporter_cuda(hip_stream);
  engine_cpu.reporter.on("data", &reporter_cpu);
  engine_cuda.reporter.on("data", &reporter_cuda);


  Reference::To<dsp::TimeSeries> in = new dsp::TimeSeries;
  Reference::To<dsp::TimeSeries> out = new dsp::TimeSeries;
  Reference::To<dsp::TimeSeries> in_gpu = new dsp::TimeSeries;
  Reference::To<dsp::TimeSeries> out_gpu = new dsp::TimeSeries;
  Reference::To<dsp::TimeSeries> out_cuda = new dsp::TimeSeries;

  Rational os_factor (4, 3);
  unsigned npart = test_shape.npart;

  util::IntegrationTestConfiguration<dsp::InverseFilterbank> config(
    os_factor, npart, test_shape.input_npol,
    test_shape.input_nchan, test_shape.output_nchan,
    test_shape.input_ndat, test_shape.overlap_pos
  );
  config.filterbank->set_pfb_dc_chan(true);
  config.filterbank->set_pfb_all_chan(true);

  config.setup (in, out);

  engine_cpu.setup(config.filterbank);
  std::vector<float *> scratch_cpu = config.allocate_scratch<dsp::Memory> ();
  engine_cpu.set_scratch(scratch_cpu[0]);
  engine_cpu.perform(
    in, out, npart
  );
  engine_cpu.finish();
  auto transfer = util::transferTimeSeries(hip_stream, device_memory);
  transfer(in, in_gpu, hipMemcpyHostToDevice);
  transfer(out, out_gpu, hipMemcpyHostToDevice);

  // config.filterbank->set_device(device_memory);
  engine_cuda.setup(config.filterbank);
  std::vector<float *> scratch_cuda = config.allocate_scratch<CUDA::DeviceMemory>(device_memory);
  engine_cuda.set_scratch(scratch_cuda[0]);
  engine_cuda.perform(
    in_gpu, out_gpu, npart
  );
  engine_cuda.finish();
  // now lets compare the two time series
  transfer(out_gpu, out_cuda, hipMemcpyDeviceToHost);

  // for (unsigned i=0; i<reporter_cpu.data_vectors.size(); i++)
  // {
  //   REQUIRE(util::allclose<float>(
  //     reporter_cpu.data_vectors[i],
  //     reporter_cuda.data_vectors[i],
  //     thresh[0], thresh[1]
  //   ));
  // }



  REQUIRE(util::allclose(out_cuda, out, thresh[0], thresh[1]));


}
