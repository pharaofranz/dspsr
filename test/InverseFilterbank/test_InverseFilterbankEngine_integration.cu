#include <vector>

#include "catch.hpp"

#include "dsp/InverseFilterbank.h"
#include "dsp/InverseFilterbankEngine.h"
#include "dsp/InverseFilterbankEngineCPU.h"
#include "dsp/InverseFilterbankEngineCUDA.h"
#include "dsp/MemoryCUDA.h"

#include "util.hpp"
#include "InverseFilterbankTestConfig.hpp"

static util::InverseFilterbank::InverseFilterbankTestConfig test_config;

void check_error (const char*);

class Reporter : public dsp::InverseFilterbank::Engine::Reporter {
public:

  Reporter (bool _iscuda = false): iscuda(_iscuda) {}

  Reporter (hipStream_t _stream): stream(_stream) { iscuda = true; }

  void operator() (float* arr, unsigned nchan, unsigned npol, unsigned ndat, unsigned ndim)
  {
    if (util::config::verbose)
    {
      std::cerr << "Reporter::operator() ("
        << arr << ", "
        << nchan << ", "
        << npol << ", "
        << ndat << ", "
        << ndim << ")"
        << std::endl;
    }
    unsigned total_size = nchan * npol * ndat * ndim;
    std::vector<float> data;
    if (iscuda) {
      data = std::vector<float>(total_size);
      float* data_ptr = data.data();
      size_t total_size_bytes = total_size * sizeof(float);
      hipError_t error;
      if (stream) {
        error = hipMemcpyAsync(data_ptr, arr, total_size_bytes, hipMemcpyDeviceToHost, stream);
        if (error != 0) {
          throw "hipMemcpyAsync error";
        }
        error = hipStreamSynchronize(stream);
      } else {
        error = hipMemcpy((void*) data_ptr, (void*) arr, total_size_bytes, hipMemcpyDeviceToHost);
        if (error != 0) {
          throw "hipMemcpy error";
        }
        error = hipDeviceSynchronize();
      }
      check_error("Reporter::operator()");
    } else {
      data = std::vector<float>(arr, arr + total_size);
    }
    data_vectors.push_back(data);
  }

  hipStream_t stream;
  bool iscuda;
  std::vector<std::vector<float>> data_vectors;

};



TEST_CASE (
  "InverseFilterbankEngineCPU and InverseFilterbankEngineCUDA produce same output",
  "[InverseFilterbankEngineCPU]"
)
{

  std::vector<float> thresh = test_config.get_thresh();
  std::vector<util::TestShape> test_shapes = test_config.get_test_vector_shapes();
  auto idx = GENERATE_COPY(range(0, (int) test_shapes.size()));
  util::TestShape test_shape = test_shapes[idx];


  void* stream = 0;
  hipStream_t hip_stream = reinterpret_cast<hipStream_t>(stream);
  CUDA::DeviceMemory* device_memory = new CUDA::DeviceMemory(hip_stream);
  CUDA::InverseFilterbankEngineCUDA engine_cuda(hip_stream);
  dsp::InverseFilterbankEngineCPU engine_cpu;

  Reporter reporter_cpu;
  Reporter reporter_cuda(hip_stream);
  engine_cpu.reporter.on("data", &reporter_cpu);
  engine_cuda.reporter.on("data", &reporter_cuda);


  Reference::To<dsp::TimeSeries> in = new dsp::TimeSeries;
  Reference::To<dsp::TimeSeries> out = new dsp::TimeSeries;
  Reference::To<dsp::TimeSeries> in_gpu = new dsp::TimeSeries;
  Reference::To<dsp::TimeSeries> out_gpu = new dsp::TimeSeries;
  Reference::To<dsp::TimeSeries> out_cuda = new dsp::TimeSeries;

  Rational os_factor (4, 3);
  unsigned npart = test_shape.npart;

  util::IntegrationTestConfiguration<dsp::InverseFilterbank> config(
    os_factor, npart, test_shape.input_npol,
    test_shape.input_nchan, test_shape.output_nchan,
    test_shape.input_ndat, test_shape.overlap_pos
  );
  config.filterbank->set_pfb_dc_chan(true);
  config.filterbank->set_pfb_all_chan(true);

  config.setup (in, out);

  engine_cpu.setup(config.filterbank);
  std::vector<float *> scratch_cpu = config.allocate_scratch<dsp::Memory> ();
  engine_cpu.set_scratch(scratch_cpu[0]);
  engine_cpu.perform(
    in, out, npart
  );
  engine_cpu.finish();
  auto transfer = util::transferTimeSeries(hip_stream, device_memory);
  transfer(in, in_gpu, hipMemcpyHostToDevice);
  transfer(out, out_gpu, hipMemcpyHostToDevice);

  // config.filterbank->set_device(device_memory);
  engine_cuda.setup(config.filterbank);
  std::vector<float *> scratch_cuda = config.allocate_scratch<CUDA::DeviceMemory>(device_memory);
  engine_cuda.set_scratch(scratch_cuda[0]);
  engine_cuda.perform(
    in_gpu, out_gpu, npart
  );
  engine_cuda.finish();
  // now lets compare the two time series
  transfer(out_gpu, out_cuda, hipMemcpyDeviceToHost);
  // std::cerr << "reporter_cpu.data_vectors.size()=" << reporter_cpu.data_vectors.size() << std::endl;
  // std::cerr << "reporter_cuda.data_vectors.size()=" << reporter_cuda.data_vectors.size() << std::endl;
  REQUIRE(util::allclose(out_cuda, out, thresh[0], thresh[1]));


}
