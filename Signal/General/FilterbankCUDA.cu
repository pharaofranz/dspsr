#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright(C) 2010 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

//#define _DEBUG 1

#include "dsp/FilterbankCUDA.h"
#include "CUFFTError.h"
#include "debug.h"

#include <hip/hip_runtime.h>

#include <iostream>
#include <assert.h>

void check_error_stream(const char*, hipStream_t);

#ifdef _DEBUG
#define CHECK_ERROR(x,y) check_error_stream(x,y)
#else
#define CHECK_ERROR(x,y)
#endif

__global__ void k_multiply(float2* d_fft, float2* kernel)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	float x = d_fft[i].x * kernel[i].x - d_fft[i].y * kernel[i].y;
	d_fft[i].y = d_fft[i].x * kernel[i].y + d_fft[i].y * kernel[i].x;
	d_fft[i].x = x;
}

__global__ void k_ncopy(float2* output_data, unsigned output_stride,
			const float2* input_data, unsigned input_stride,
			unsigned to_copy)
{
	output_data += blockIdx.y * output_stride;
	input_data += blockIdx.y * input_stride;
	unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < to_copy) {
		output_data[index] = input_data[index];
	}
}

using namespace std;

void FilterbankEngineCUDA::setup(dsp::Filterbank* filterbank)
{
	// the CUDA engine does not maintain/compute the passband
	filterbank->set_passband(NULL);
	//
	_frequencyResolution = filterbank->get_freq_res();
	_nChannelSubbands = filterbank->get_nchan_subband();
	_realToComplex =(filterbank->get_input()->get_state() == Signal::Nyquist);
	DEBUG("FilterbankEngineCUDA::setup _nChannelSubbands=" << _nChannelSubbands
		<< " _frequencyResolution=" << _frequencyResolution);
	DEBUG("FilterbankEngineCUDA::setup scratch=" << scratch);
	hipfftResult result;
	if(_realToComplex) {
		DEBUG("FilterbankEngineCUDA::setup plan size=" << _frequencyResolution*_nChannelSubbands*2);
		result = hipfftPlan1d(&_planForward, _frequencyResolution*_nChannelSubbands*2, HIPFFT_R2C, 1);
		if(result != HIPFFT_SUCCESS) {
			throw CUFFTError(	result, "FilterbankEngineCUDA::setup",
						"hipfftPlan1d(_planForward, HIPFFT_R2C)");
		}
	} else {
		DEBUG("FilterbankEngineCUDA::setup plan size=" << _frequencyResolution*_nChannelSubbands);
		result = hipfftPlan1d(&_planForward, _frequencyResolution*_nChannelSubbands, HIPFFT_C2C, 1);
		if(result != HIPFFT_SUCCESS) {
			throw CUFFTError(	result, "FilterbankEngineCUDA::setup",
						"hipfftPlan1d(_planForward, HIPFFT_C2C)");
		}
	}
	DEBUG("FilterbankEngineCUDA::setup setting _stream=" << _stream);
	result = hipfftSetStream(_planForward, _stream);
	if(result != HIPFFT_SUCCESS) {
		throw CUFFTError(	result, "FilterbankEngineCUDA::setup",
					"hipfftSetStream(_planForward)");
	}
	DEBUG("FilterbankEngineCUDA::setup fwd FFT plan set");
	if(_frequencyResolution > 1) {
		result = hipfftPlan1d(&_planBackward, _frequencyResolution, HIPFFT_C2C, _nChannelSubbands);
		if(result != HIPFFT_SUCCESS) {
			throw CUFFTError(	result, "FilterbankEngineCUDA::setup",
						 "hipfftPlan1d(_planBackward)");
		}
		result = hipfftSetStream(_planBackward, _stream);
		if(result != HIPFFT_SUCCESS) {
			throw CUFFTError(	result, "FilterbankEngineCUDA::setup",
						"hipfftSetStream(_planBackward)");
		}
		DEBUG("FilterbankEngineCUDA::setup bwd FFT plan set");
	}
	_nKeep = _frequencyResolution;
	_multiply.init();
	_multiply.set_nelement(_nChannelSubbands * _frequencyResolution);
	if(filterbank->has_response()) {
		const dsp::Response* response = filterbank->get_response();
		unsigned nchan = response->get_nchan();
		unsigned ndat = response->get_ndat();
		unsigned ndim = response->get_ndim();
		assert( nchan == filterbank->get_nchan() );
		assert( ndat == _frequencyResolution );
		assert( ndim == 2 ); // complex
		unsigned mem_size = nchan * ndat * ndim * sizeof(hipfftReal);	
		// allocate space for the convolution kernel
		hipMalloc((void**)&_convolutionKernel, mem_size);
		_nFilterPosition = response->get_impulse_pos();
		unsigned nfilt_tot = _nFilterPosition + response->get_impulse_neg();
		// points kept from each small fft
		_nKeep = _frequencyResolution - nfilt_tot;
		// copy the kernel accross
		const float* kernel = filterbank->get_response()->get_datptr(0,0);
		if(_stream) {
			hipMemcpyAsync(_convolutionKernel, kernel, mem_size, hipMemcpyHostToDevice, _stream);
		} else {
			hipMemcpy(_convolutionKernel, kernel, mem_size, hipMemcpyHostToDevice);
		}
	}
}

void FilterbankEngineCUDA::set_scratch(float * _scratch)
{
	scratch = _scratch;
}

void FilterbankEngineCUDA::finish()
{
	check_error_stream("FilterbankEngineCUDA::finish", _stream);
}

void FilterbankEngineCUDA::perform(	const dsp::TimeSeries * in, dsp::TimeSeries * out,
					uint64_t npart, const uint64_t in_step, const uint64_t out_step)
{
	verbose = dsp::Operation::record_time || dsp::Operation::verbose;
	//
	const unsigned npol = in->get_npol();
	const unsigned input_nchan = in->get_nchan();
	const unsigned output_nchan = out->get_nchan();
	// counters
	unsigned ipol, ichan;
	uint64_t ipart;
	// offsets into input and output
	uint64_t in_offset, out_offset;
	DEBUG("FilterbankEngineCUDA::perform _stream=" << _stream);
	// GPU scratch space
	DEBUG("FilterbankEngineCUDA::perform scratch=" << scratch);
	float2 *cscratch = (float2 *)scratch;
	//
	hipfftResult result;
	float *output_ptr;
	float *input_ptr;
	uint64_t output_span;
	//
	DEBUG("FilterbankEngineCUDA::perform input_nchan=" << input_nchan);
	DEBUG("FilterbankEngineCUDA::perform npol=" << npol);
	DEBUG("FilterbankEngineCUDA::perform npart=" << npart);
	DEBUG("FilterbankEngineCUDA::perform _nKeep=" << _nKeep);
	DEBUG("FilterbankEngineCUDA::perform in_step=" << in_step);
	DEBUG("FilterbankEngineCUDA::perform out_step=" << out_step);
	for(ichan = 0; ichan < input_nchan; ichan++) {
		for(ipol = 0; ipol < npol; ipol++) {
			for(ipart = 0; ipart < npart; ipart++) {
				DEBUG("FilterbankEngineCUDA::perform ipart " << ipart << " of " << npart);
				in_offset = ipart * in_step;
				out_offset = ipart * out_step;
				DEBUG("FilterbankEngineCUDA::perform offsets in=" << in_offset << " out=" << out_offset);
				input_ptr = const_cast<float *>(in->get_datptr(ichan, ipol)) + in_offset;
				DEBUG("FilterbankEngineCUDA::perform FORWARD FFT inptr=" << input_ptr << " outptr=" << cscratch);
				if(_realToComplex) {
					result = hipfftExecR2C(_planForward, input_ptr, cscratch);
					if(result != HIPFFT_SUCCESS) {
						throw CUFFTError(result, "FilterbankEngineCUDA::perform", "hipfftExecR2C");
					}
					CHECK_ERROR("FilterbankEngineCUDA::perform hipfftExecR2C FORWARD", _stream);
				} else {
					float2 *cin = (float2 *)input_ptr;
					result = hipfftExecC2C(_planForward, cin, cscratch, HIPFFT_FORWARD);
					if(result != HIPFFT_SUCCESS) {
						throw CUFFTError(result, "FilterbankEngineCUDA::perform", "hipfftExecC2C");
					}
					CHECK_ERROR("FilterbankEngineCUDA::perform hipfftExecC2C FORWARD", _stream);
				}
				if(_convolutionKernel) {
					// complex numbers offset(_convolutionKernel is float2*)
					unsigned offset = ichan * _nChannelSubbands * _frequencyResolution;
					DEBUG("FilterbankEngineCUDA::perform _multiply dedipersion kernel _stream=" << _stream);
					k_multiply<<<_multiply.get_nblock(), _multiply.get_nthread(), 0, _stream>>>(cscratch, _convolutionKernel + offset);
					CHECK_ERROR("FilterbankEngineCUDA::perform _multiply", _stream);
				}
				if(_planBackward) {
					DEBUG("FilterbankEngineCUDA::perform BACKWARD FFT");
					result = hipfftExecC2C(_planBackward, cscratch, cscratch, HIPFFT_BACKWARD);
					if(result != HIPFFT_SUCCESS) {
						throw CUFFTError(result, "FilterbankEngineCUDA::perform", "hipfftExecC2C(inverse)");
					}
					CHECK_ERROR("FilterbankEngineCUDA::perform hipfftExecC2C BACKWARD", _stream);
				}
				if(out) {
					output_ptr = out->get_datptr(ichan * _nChannelSubbands, ipol) + out_offset;
					output_span = out->get_datptr(ichan * _nChannelSubbands + 1, ipol) - out->get_datptr(ichan * _nChannelSubbands, ipol);
					//
					const float2* input = cscratch + _nFilterPosition;
					unsigned input_stride = _frequencyResolution;
					unsigned to_copy = _nKeep;
					{
						dim3 threads;
						threads.x = _multiply.get_nthread();
						//
						dim3 blocks;
						blocks.x = _nKeep / threads.x;
						if(_nKeep % threads.x) {
							blocks.x++;
						}
						blocks.y = _nChannelSubbands;
						// divide by two for complex data
						float2 *output_base = (float2 *)output_ptr;
						unsigned output_stride = output_span / 2;
						DEBUG("FilterbankEngineCUDA::perform output base=" << output_base << " stride=" << output_stride);
						DEBUG("FilterbankEngineCUDA::perform input base=" << input << " stride=" << input_stride);
						DEBUG("FilterbankEngineCUDA::perform to_copy=" << to_copy);
						k_ncopy<<<blocks, threads, 0, _stream>>>(output_base, output_stride,
											 input, input_stride, to_copy);
						CHECK_ERROR("FilterbankEngineCUDA::perform ncopy", _stream);
					}
				} // if not benchmarking
			} // for each part
		} // for each polarization
	} // for each channel
	if(verbose) {
		check_error_stream("FilterbankEngineCUDA::perform", _stream);
	}
}
