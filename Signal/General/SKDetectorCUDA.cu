#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2016 by Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/


#include "dsp/SKDetectorCUDA.h"

#include <iostream>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#include <hip/hip_complex.h>

#define FULL_MASK 0xffffffff

//#define _DEBUG 1

// TODO consider having schan / echan in mask represented by values other than 0, 1

using namespace std;

void check_error (const char*);

CUDA::SKDetectorEngine::SKDetectorEngine (dsp::Memory * memory)
{
  device_memory = dynamic_cast<CUDA::DeviceMemory *>(memory);
  stream = device_memory->get_stream();

  estimates_host = new dsp::TimeSeries();
  zapmask_host = new dsp::BitSeries();

  pinned_memory  = new PinnedMemory ();
  estimates_host->set_memory ((dsp::Memory *) pinned_memory);
  zapmask_host->set_memory ((dsp::Memory *) pinned_memory);

  transfer_estimates = new dsp::TransferCUDA (stream);
  transfer_estimates->set_kind (hipMemcpyDeviceToHost);
  transfer_estimates->set_output( estimates_host );

  transfer_zapmask = new dsp::TransferBitSeriesCUDA (stream);
  transfer_zapmask->set_kind (hipMemcpyDeviceToHost);
  transfer_zapmask->set_output( zapmask_host );
}

void CUDA::SKDetectorEngine::setup ()
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::SKDetectorEngine::setup ()" << endl;

  // determine GPU capabilities
  int device = 0;
  hipGetDevice(&device);
  struct hipDeviceProp_t device_properties;
  hipGetDeviceProperties (&device_properties, device);
  max_threads_per_block = device_properties.maxThreadsPerBlock;
}


// faster kernel for npol=1
__global__ void detect_one_pol (const float * indat, unsigned char * outdat, uint64_t nval, float upper, float lower)
{
  unsigned idat  = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idat < nval)
  {
    float V = indat[idat];
    if (V < lower || V > upper)
      outdat[idat] = 1;
  }
}

__global__ void detect_two_pol (const float2 * indat, unsigned char * outdat, uint64_t nval, float upper, float lower)
{
  unsigned idat  = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idat < nval)
  {
    const float2 V = indat[idat];
    if (V.x < lower || V.x > upper || V.y < lower || V.y > upper)
    {
      outdat[idat] = 1;
    }
  }
}


// detect SK limits for N polarisations
__global__ void detect_one_sample (const float * indat, unsigned char * outdat, uint64_t nval, float upper, float lower, unsigned npol)
{
  unsigned idat  = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (idat < nval)
  {
    unsigned zap = 0;
    float V;

    for (int ipol=0; ipol<npol; ipol++)
    {
      V = indat[(idat * npol) + ipol];
      if (V < lower || V > upper)
      {
        zap = 1;
      }
    }
    if (zap)
      outdat[idat] = 1;
  }
}

void CUDA::SKDetectorEngine::detect_ft (const dsp::TimeSeries* input,
      dsp::BitSeries* output, float upper_thresh, float lower_thresh)
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::SKDetectorEngine::detect_ft()" << endl;

  const unsigned nchan = input->get_nchan();
  const unsigned npol  = input->get_npol();
  const int64_t  ndat  = input->get_ndat();

  const float * indat    = input->get_dattfp();   // TFP
  unsigned char * outdat = output->get_datptr();  // TFP also!

  uint64_t nval   = nchan * ndat;
  uint64_t nblocks  = nval / max_threads_per_block;
  if (nval % max_threads_per_block)
    nblocks++;

  dim3 threads (max_threads_per_block);
  dim3 blocks (nblocks);

  if (dsp::Operation::verbose)
  {
    cerr << "CUDA::SKDetectorEngine::detect_ft nval=" << nval << " nblocks=" << nblocks << " max_threads_per_block=" << max_threads_per_block << endl;
    cerr << "CUDA::SKDetectorEngine::detect_ft thresholds [" << lower_thresh << " - " << upper_thresh << "]" << endl;
    cerr << "CUDA::SKDetectorEngine::detect_ft npol=" << npol << endl;
  }

  if (npol == 1)
    detect_one_pol<<<blocks,threads,npol,stream>>> (indat, outdat, nval, upper_thresh, lower_thresh);
  else if (npol == 2)
    detect_two_pol<<<blocks,threads,npol,stream>>> ((const float2 *) indat, outdat, nval, upper_thresh, lower_thresh);
  else
    detect_one_sample<<<blocks,threads,npol,stream>>> (indat, outdat, nval, upper_thresh, lower_thresh, npol);

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error( "CUDA::SKDetectorEngine::detect_ft detect_one_xxx" );

#ifdef _DEBUG
  int sum = count_mask(output);
  cerr << "CUDA::SKDetectorEngine::detect_ft sum now " << sum << endl;
#endif
}

__device__ float2 warp_reduce_sum (float2 val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) {
    #if (__CUDACC_VER_MAJOR__>= 9)
    val.x += __shfl_down_sync(FULL_MASK, val.x, offset);
    val.y += __shfl_down_sync(FULL_MASK, val.y, offset);
    #else
    val.x += __shfl_down (val.x, offset);
    val.y += __shfl_down (val.y, offset);
    #endif
  }
  return val;
}

__device__ float3 warp_reduce_sum (float3 val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) {
    #if (__CUDACC_VER_MAJOR__>= 9)
    val.x += __shfl_down_sync(FULL_MASK, val.x, offset);
    val.y += __shfl_down_sync(FULL_MASK, val.y, offset);
    val.z += __shfl_down_sync(FULL_MASK, val.z, offset);
    #else
    val.x += __shfl_down (val.x, offset);
    val.y += __shfl_down (val.y, offset);
    val.z += __shfl_down (val.z, offset);
    #endif
  }
  return val;
}





// each block reads 1 time sample, all channels/pols
// then do a block-wide sum

// input data are stored TFP, 1 warp per time sample, 32 warps / block to sum across channels
__global__ void reduce_sum_fscr_1pol (const float * input, unsigned char * out,
                                      const unsigned nchan, float lower, float upper,
                                      unsigned schan, unsigned echan)
{
  extern __shared__ float sdata[];

  unsigned idat = blockIdx.x;
  const float * in = input + (idat * nchan);

  float sum = 0;
  for (unsigned ichan=threadIdx.x; ichan<nchan; ichan+=blockDim.x)
  {
    if (ichan >= schan && ichan < echan)
      sum += in[ichan];
  }

  sdata[threadIdx.x] = sum;
  __syncthreads();

  // now do a block wide sum across all threads
  int last_offset = blockDim.x / 2 ;
  for (int offset = last_offset; offset > 0;  offset >>= 1)
  {
    if (threadIdx.x < offset)
      sdata[threadIdx.x] += sdata[threadIdx.x + offset];

    __syncthreads();
  }

  if (threadIdx.x == 0)
  {
    float val = sdata[0] / float((echan - schan) + 1);
    if (val < lower || val > upper)
      out[idat] = 1;
  }
}

//! blockDim.x is nchan, so threadIdx.x is ichan
//! gridDim.x is input->get_ndat(), or npart, os blockIdx.x is ipart
//! input is TFP (npart, nchan, npol)
//! out is TFP (npart, nchan, 1)
__global__ void reduce_sum_fscr_2pol (
  const float2 * input, unsigned char * out,
  const unsigned nchan, const float mu2, const unsigned std_devs,
  const unsigned schan, const unsigned echan
)
{
  extern __shared__ float3 sdata3[]; // we have nchan * (npol + 1) * sizeof(float) available bytes

  // idat = blockIdx.x
  // use float 2 because input is TFP, meaning we can bundle polarizations
  // as if they were complex number
  const float2 * in = input + (blockIdx.x * nchan);

  float3 sum = make_float3(0, 0, 0);
  for (unsigned ichan=threadIdx.x; ichan<nchan; ichan+=blockDim.x)
  {
    if (ichan >= schan && ichan < echan && out[blockIdx.x * nchan + ichan] == 0) {
      sum.x += in[ichan].x;
      sum.y += in[ichan].y;
      sum.z += 1;
    }
  }

  sum = warp_reduce_sum(sum);

  unsigned warp_idx = threadIdx.x % 32;
  unsigned warp_num = threadIdx.x / 32;

  if (warp_idx == 0) {
    sdata3[warp_num] = sum;
  }
  __syncthreads();

  if (warp_num == 0) {
    sum = sdata3[warp_idx];
    sum = warp_reduce_sum(sum);

    if (warp_idx == 0) {
      float sk_avg_cnt = sum.z;
      float one_sigma_idat = sqrtf(mu2 / (float) sk_avg_cnt);
      float p0 = sum.x / sk_avg_cnt;
      float p1 = sum.y / sk_avg_cnt;
      float upper = 1 + ((1+std_devs) * one_sigma_idat);
      float lower = 1 - ((1+std_devs) * one_sigma_idat);
      printf("reduce_sum_fscr_2pol: p0=%f, p1=%f, lower=%f, upper=%f, sk_avg_cnt=%f, pol0 sum=%f, pol1 sum=%f\n", p0, p1, lower, upper, sk_avg_cnt, p0*sk_avg_cnt, p1*sk_avg_cnt);
      if (p0 < lower || p0 > upper || p1 < lower || p1 > upper) {
        for (unsigned ichan=0; ichan<nchan; ichan+=1) {
          out[blockIdx.x * nchan + ichan] = 1;
        }
      }
    }
  }
  // sdata3[threadIdx.x] = warp_reduce_sum(sdata3[threadIdx.x]);

  // now do a block wide sum across all threads
  // int last_offset = blockDim.x / 2;
  // for (int offset = last_offset; offset > 0;  offset >>= 1) // bitshift down by one
  // {
  //   if (threadIdx.x < offset) {
  //     sdata3[threadIdx.x].x += sdata3[threadIdx.x + offset].x;
  //     sdata3[threadIdx.x].y += sdata3[threadIdx.x + offset].y;
  //     sdata3[threadIdx.x].z += sdata3[threadIdx.x + offset].z;
  //   }
  //   __syncthreads();
  // }
}

// schan is the start channel and echan is the end channel. Together these
// define a range of channels that will be zapped.
// input is the TFP ordered SK estimates, of size (npart, nchan, npol)
// output is the TFP ordered zapmask, of size (npart, nchan, 1)
// Here, npart is the original TimeSeries input ndat divided by ``M``
void CUDA::SKDetectorEngine::detect_fscr (
  const dsp::TimeSeries* input, dsp::BitSeries* output,
  const float mu2, const unsigned std_devs,
  unsigned schan, unsigned echan)
{
  if (dsp::Operation::verbose) {
    std::cerr << "CUDA::SKDetectorEngine::detect_fscr()" << std::endl;
  }

  const unsigned nchan = input->get_nchan();
  const unsigned npol = input->get_npol();
  const int64_t ndat = input->get_ndat();

  const unsigned nblocks = ndat;
  unsigned nthreads = max_threads_per_block;
  if (nchan < nthreads)
    nthreads = nchan;
  const size_t shared_bytes = nthreads * (npol + 1) * sizeof(float);

  // indat is the SK estimatesestimates
  const float * indat    = input->get_dattfp();

  // outdat is the bitmask
  unsigned char * outdat = output->get_datptr();
  std::cerr << "CUDA::SKDetectorEngine::detect_fscr:"
    << " output->get_ndat()=" << output->get_ndat()
    << " output->get_npol()=" << output->get_npol()
    << " output->get_nchan()=" << output->get_nchan() << std::endl;
  // if (dsp::Operation::verbose)
  // {
    std::cerr << "CUDA::SKDetectorEngine::detect_fscr nchan=" << nchan << " ndat=" << ndat << std::endl;
    std::cerr << "CUDA::SKDetectorEngine::detect_fscr nblocks=" << nblocks << " nthreads=" << nthreads << " shared_bytes=" << shared_bytes << std::endl;
    // std::cerr << "CUDA::SKDetectorEngine::detect_fscr thresholds [" << lower << " - " << upper << "]" << std::endl;
  // }

  // if (npol == 1) {
  //   reduce_sum_fscr_1pol<<<nblocks, nthreads,s hared_bytes, stream>>>(
  //     indat, outdat, nchan, lower, upper, schan, echan);
  // } else {
    reduce_sum_fscr_2pol<<<nblocks, nthreads, shared_bytes, stream>>>(
      (float2*) indat, outdat, nchan, mu2, std_devs, schan, echan);
  // }

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error( "CUDA::SKDetectorEngine::detect_fscr_element" );

#ifdef _DEBUG
  int sum = count_mask(output);
  cerr << "CUDA::SKDetectorEngine::detect_fscr mask_sum=" << sum << endl;
#endif

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error( "CUDA::SKDetectorEngine::detect detect_fscr" );
}

// nval is output->get_ndat() * nchan
// indat is TFP ordered
// indat is (1, nchan, npol)
// outdat is (ndat, nchan, npol)
__global__ void detect_tscr_element (
  const float * indat,
  unsigned char * outdat,
  uint64_t nval,
  float upper,
  float lower,
  unsigned npol,
  unsigned nchan
)
{

  extern __shared__ char sk_tscr[];

  unsigned int idat  = (blockIdx.x * blockDim.x + threadIdx.x);
  bool all_pol_in_thresh;
  if (idat ==0) {
    printf("detect_tscr_element: npol=%u, nchan=%u\n", npol, nchan);
  }
  if (idat < nval)
  {
    // const unsigned nchanpol = nchan * npol;
    // const unsigned ichanpol = idat % nchanpol;

    // first nchan threads to fill shared mem with the tscr SK estimates for each chan & pol (TFP)

    // if (threadIdx.x < nchan)
    // {
    //   // sk_tscr[threadIdx.x] = (char) (indat[threadIdx.x] > upper || indat[threadIdx.x] < lower);
    //   all_pol_in_thresh = false;
    //   for (unsigned ipol=0; ipol<npol; ipol++) {
    //     all_pol_in_thresh = all_pol_in_thresh || (indat[threadIdx.x*npol + ipol] > upper || indat[threadIdx.x*npol + ipol] < lower);
    //   }
    //   sk_tscr[threadIdx.x] = (char) all_pol_in_thresh;
    // }
    // __syncthreads();
    // outdat[idat/npol] = sk_tscr[ichanpol];

    if (threadIdx.x < nchan)
    {
      all_pol_in_thresh = false;
      for (unsigned ipol=0; ipol<npol; ipol++) {
        all_pol_in_thresh = (all_pol_in_thresh ||
          (indat[threadIdx.x*npol + ipol] > upper || indat[threadIdx.x*npol + ipol] < lower));
      }
      sk_tscr[threadIdx.x] = (char) all_pol_in_thresh;
    }
    __syncthreads();
    outdat[idat] = sk_tscr[idat % nchan];

  }
}


void CUDA::SKDetectorEngine::detect_tscr (const dsp::TimeSeries* input,
      const dsp::TimeSeries* input_tscr, dsp::BitSeries* output,
      float upper_thresh, float lower_thresh)
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::SKDetectorEngine::detect_tscr()" << endl;
  const unsigned nchan   = input->get_nchan();
  const unsigned npol    = input->get_npol();
  const int64_t ndat     = output->get_ndat();

  // indat is the tscr mask [nchan vals]
  const float * indat    = input_tscr->get_dattfp();

  // outdat is the bitmask
  unsigned char * outdat = output->get_datptr();

  // this kernel is indexed on output rather than input
  const uint64_t nval = ndat * nchan;
  uint64_t nblocks  = nval / max_threads_per_block;
  if (nval % max_threads_per_block)
    nblocks++;

  dim3 threads (max_threads_per_block);
  dim3 blocks (nblocks);
  unsigned shared_bytes = nchan*npol*sizeof(char);

  // if (dsp::Operation::verbose)
  cerr << "CUDA::SKDetectorEngine::detect_tscr_element ndat=" << ndat
      << " npol=" << npol
       << " nchan=" << nchan << " nval=" << nval
       << " max_threads=" << max_threads_per_block
       << " nblocks=" << nblocks << endl;

  detect_tscr_element<<<blocks,threads,shared_bytes,stream>>>(
    indat, outdat, nval, upper_thresh, lower_thresh, npol, nchan);

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error( "CUDA::SKDetectorEngine::detect_tscr_element" );

#ifdef _DEBUG
  int sum = count_mask(output);
  cerr << "CUDA::SKDetectorEngine::detect_tscr mask_sum=" << sum << endl;
#endif
}


void CUDA::SKDetectorEngine::reset_mask (dsp::BitSeries* output)
{
  unsigned nchan         = output->get_nchan();
  int64_t ndat           = output->get_ndat();
  unsigned char * outdat = output->get_datptr();

  size_t nbytes = nchan * ndat;

  hipError_t error = hipMemsetAsync (outdat, 0, nbytes, stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::SKDetectorEngine::reset_mask ",
                 "hipMemset (%p, 0, %u): %s", outdat, nbytes,
                 hipGetErrorString (error));
#ifdef _DEBUG
  int sum = count_mask(output);
  cerr << "CUDA::SKDetectorEngine::reset_mask sum now " << sum << endl;
#endif
}

int CUDA::SKDetectorEngine::count_mask (const dsp::BitSeries* output)
{
  unsigned char * outdat = const_cast<unsigned char *>(output->get_datptr());
  const unsigned nchan   = output->get_nchan();
  const int64_t ndat     = output->get_ndat();
  int sum = 0;
/*
  const uint64_t nval    = (uint64_t) ndat * nchan;
  hipStreamSynchronize(stream);
  thrust::device_ptr<unsigned char> d = thrust::device_pointer_cast(outdat);
  int sum = thrust::reduce(thrust::cuda::par.on(stream), d, d+nval, (int) 0, thrust::plus<int>());
  hipStreamSynchronize(stream);
*/

  return sum;
}

float * CUDA::SKDetectorEngine::get_estimates (const dsp::TimeSeries * input)
{
  transfer_estimates->set_input (input);
  transfer_estimates->operate ();
  hipStreamSynchronize (stream);
  return estimates_host->get_dattfp();
}

unsigned char * CUDA::SKDetectorEngine::get_zapmask (const dsp::BitSeries * input)
{
  transfer_zapmask->set_input (input);
  transfer_zapmask->operate ();
  hipStreamSynchronize (stream);
  return zapmask_host->get_datptr();
}
