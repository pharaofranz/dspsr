#include "hip/hip_runtime.h"
//-*-C++-*-
/***************************************************************************
 *
 *   Copyright (C) 2019 by Willem van Straten, Andrew Jameson and Dean Shaff
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/
#include <cstdio>
#include <vector>

#include <hip/hip_runtime.h>

#include "CUFFTError.h"
#include "dsp/InverseFilterbankEngineCUDA.h"


/*!
 * Kernel for multiplying a time domain array by an apodization kernel,
 * removing any overlap discard regions in the process.
 * \method k_apodization_overlap
 * \param t_in the input data array pointer. The shape of the array should be
 *    (nchan, ndat)
 * \param apodization the apodization kernel
 * \param t_out the output data array pointer
 * \param discard the overlap discard region, in *complex samples*
 * \param ndat the number of time samples in t_in
 * \param nchan the number of channels in t_in
 */
__global__ void k_apodization_overlap (
  float2* t_in,
  float2* apodization,
  float2* t_out,
  int discard,
  int ndat,
  int nchan
)
{
  int total_size_x = blockDim.x * gridDim.x;
  int total_size_y = blockDim.y * gridDim.y;

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int apod_ndat = ndat - 2*discard;
  // printf("ndat=%d, nchan=%d, idx=%d, total_size_x=%d, total_size_y=%d\n", ndat, nchan, idx, total_size_x, total_size_y);

  // make sure we're not trying to access channels that don't exist
  if (blockIdx.y > nchan) {
    return;
  }
  // ignore top of overlap region
  if (idx > (ndat - discard)) {
    return;
  }

  for (int ichan=blockIdx.y; ichan < nchan; ichan += total_size_y) {
    for (int idat=idx; idat < (ndat - discard); idat += total_size_x) {
      // printf("ichan=%d, nchan=%d, idat=%d, t_in=(%f, %f), apodization=(%f, %f)\n",
      //   ichan, nchan, idat, t_in[ichan*ndat + idat].x, t_in[ichan*ndat + idat].y,
      //   apodization[idat-discard].x, apodization[idat-discard].y);
      // ignore bottom of overlap region
      if (idat < discard) {
        continue;
      }
      t_out[ichan*apod_ndat + (idat - discard)] = hipCmulf(apodization[idat - discard], t_in[ichan*ndat + idat]);
    }
  }
}


/*!
 * Kernel for multiplying a Response or ResponseProduct's internal
 * buffer by result of forward FFTs. After multiplying, this stitches
 * output spectra together.
 * \method k_response_stitch
 * \param f_in the frequency domain input data pointer.
 *    Dimensions are (npol*input_nchan, input_ndat).
 *    Here, input_ndat is equal to the size of the forward FFT.
 * \param response the Response or ResponseProduct's buffer
 * \param f_out the frequency domain output data pointer.
 *    Dimensions are (npol*1, output_ndat).
 *    Here, output_ndat is equal to the size of the backward FFT,
 *    which is in turn equal to input_nchan * input_ndat normalized by
 *    the oversampling factor.
 * \param oversampled_discard the number of *complex samples* to discard
 *    from either side of the input spectra.
 */
__global__ void k_response_stitch (
  float2* f_in,
  float2* response,
  float2* f_out,
  int oversampled_discard
)
{

}


/*!
 * Kernel for discarding overlap regions on output time domain data.
 * \method k_overlap
 * \param t_in input time domain pointer
 * \param t_out output time domain pointer
 * \param discard discard region on output data.
 */
__global__ void k_overlap (
  float2* t_in,
  float2* t_out,
  int discard
)
{

}

CUDA::InverseFilterbankEngineCUDA::InverseFilterbankEngineCUDA (hipStream_t _stream)
{
  stream = _stream;

  input_fft_length = 0;
  fft_plans_setup = false;
  response = nullptr;
  fft_window = nullptr;

  pfb_dc_chan = 0;
  pfb_all_chan = 0;
  verbose = dsp::Observation::verbose;

  hipfftHandle plans[] = {forward, backward};
  int nplans = sizeof(plans) / sizeof(plans[0]);
  hipfftResult result;
  for (int i=0; i<nplans; i++) {
    result = hipfftCreate (&plans[i]);
    if (result != HIPFFT_SUCCESS) {
      throw CUFFTError (
        result,
        "CUDA::InverseFilterbankEngineCUDA::InverseFilterbankEngineCUDA",
        "hipfftCreate");
    }
  }
}

CUDA::InverseFilterbankEngineCUDA::~InverseFilterbankEngineCUDA ()
{
  hipfftHandle plans[] = {forward, backward};
  int nplans = sizeof(plans) / sizeof(plans[0]);

  hipfftResult result;
  for (int i=0; i<nplans; i++) {
    result = hipfftDestroy (plans[i]);
    if (result != HIPFFT_SUCCESS) {
      throw CUFFTError (
        result,
        "CUDA::InverseFilterbankEngineCUDA::InverseFilterbankEngineCUDA",
        "hipfftDestroy");
    }
  }
}

void CUDA::InverseFilterbankEngineCUDA::setup (dsp::InverseFilterbank* filterbank)
{
  if (filterbank->get_input()->get_state() == Signal::Nyquist) {
    type_forward = HIPFFT_R2C;
  } else {
    type_forward = HIPFFT_C2C;
  }
}

double CUDA::InverseFilterbankEngineCUDA::setup_fft_plans (dsp::InverseFilterbank* filterbank)
{
  // taken from ConvolutionCUDA engine.
  if (dsp::Operation::verbose) {
    std::cerr << "CUDA::InverseFilterbankEngineCUDA::setup_fft_plans"
      << " input_fft_length=" << input_fft_length
      << " output_fft_length=" << output_fft_length << std::endl;
  }

  // setup forward plan
  hipfftResult result = hipfftPlan1d (&forward, input_fft_length, type_forward, 1);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::InverseFilterbankEngineCUDA::setup_fft_plans",
                      "hipfftPlan1d(forward)");

  result = hipfftSetStream (forward, stream);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::InverseFilterbankEngineCUDA::setup_fft_plans",
          "hipfftSetStream(forward)");

  // setup backward plan
  result = hipfftPlan1d (&backward, output_fft_length, HIPFFT_C2C, 1);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::InverseFilterbankEngineCUDA::setup_fft_plans",
                      "hipfftPlan1d(backward)");

  result = hipfftSetStream (backward, stream);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::InverseFilterbankEngineCUDA::setup_fft_plans",
                      "hipfftSetStream(backward)");

  // size_t buffer_size = output_fft_length * sizeof (hipfftComplex);
  // hipError_t error = hipMalloc ((void **) &buf, buffer_size);
  // if (error != hipSuccess)
  //   throw Error (FailedCall, "CUDA::InverseFilterbankEngineCUDA::setup_fft_plans",
  //                "hipMalloc(%x, %u): %s", &buf, buffer_size,
  //                hipGetErrorString (error));

  // Compute FFT scale factors
  scalefac = 1.0;
  if (FTransform::get_norm() == FTransform::unnormalized) {
    scalefac = pow(double(output_fft_length), 2);
    scalefac *= pow(filterbank->get_oversampling_factor().doubleValue(), 2);
  }
  fft_plans_setup = true;
  if (verbose) {
    std::cerr << "dsp::InverseFilterbankEngineCPU::setup_fft_plans: scalefac=" << scalefac << std::endl;
  }

  return scalefac;
}

void CUDA::InverseFilterbankEngineCUDA::set_scratch (float* )
{ }

void CUDA::InverseFilterbankEngineCUDA::perform (const dsp::TimeSeries* in, dsp::TimeSeries* out,
              uint64_t npart, uint64_t in_step, uint64_t out_step)
{ }

void CUDA::InverseFilterbankEngineCUDA::finish ()
{ }

//! This method is static
void CUDA::InverseFilterbankEngineCUDA::apply_k_apodization_overlap (
  std::complex<float>* in,
  std::complex<float>* apodization,
  std::complex<float>* out,
  int discard,
  int ndat,
  int nchan)
{
  float2* in_device;
  float2* apod_device;
  float2* out_device;

  size_t sz = sizeof(float2);
  int ndat_apod = (ndat - 2*discard);

  hipMalloc((void **) &in_device, ndat*nchan*sz);
  hipMalloc((void **) &apod_device, ndat_apod*sz);
  hipMalloc((void **) &out_device, ndat_apod*nchan*sz);

  hipMemcpy(
    in_device, (float2*) in, ndat*nchan*sz, hipMemcpyHostToDevice);
  hipMemcpy(
    apod_device, (float2*) apodization, ndat_apod*sz, hipMemcpyHostToDevice);

  // 10 is sort of arbitrary here.
  dim3 grid (10, nchan, 1);
  dim3 threads (64, 1, 1);


  k_apodization_overlap<<<grid, threads>>>(
    in_device, apod_device, out_device, discard, ndat, nchan);

  hipMemcpy((float2*) out, out_device, ndat_apod*nchan*sz, hipMemcpyDeviceToHost);

  hipFree(in_device);
  hipFree(apod_device);
  hipFree(out_device);
}
