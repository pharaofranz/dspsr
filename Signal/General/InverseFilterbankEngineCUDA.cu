#include "hip/hip_runtime.h"
//-*-C++-*-
/***************************************************************************
 *
 *   Copyright (C) 2019 by Willem van Straten, Andrew Jameson and Dean Shaff
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/
#include <cstdio>
#include <vector>
#include <complex>

#include "CUFFTError.h"
#include "dsp/InverseFilterbankEngineCUDA.h"

void check_error (const char*);

/*!
 * Kernel for removing any overlap discard regions, optionally multiplying
 * by a response kernel in the process.
 * \method k_overlap_discard
 * \param t_in the input data array pointer. The shape of the array should be
 *    (nchan, ndat)
 * \param apodization the apodization kernel
 * \param t_out the output data array pointer
 * \param discard the overlap discard region, in *complex samples*
 * \param ndat the number of time samples in t_in
 * \param nchan the number of channels in t_in
 */
__global__ void k_overlap_discard (
  float2* t_in,
  float2* resp,
  float2* t_out,
  int discard,
  int npart,
  int npol,
  int nchan,
  int ndat
)
{
  int total_size_x = blockDim.x * gridDim.x; // for ndat
  int total_size_y = blockDim.y * gridDim.y; // for nchan
  int total_size_z = blockDim.z * gridDim.z; // for npart and npol
  int npol_incr = total_size_z <= npol ? 1: npol;
  int npart_incr = total_size_z/npol == 0 ? 1: total_size_z/npol;

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int idz = blockIdx.z*blockDim.z + threadIdx.z;

  // if (npol == 2){
  //   printf("total_size_z=%d, npol_incr=%d, idz=%d, idz%%npol=%d, idz/npol=%d\n",
  //     total_size_z, npol_incr, idz, idz%npol, idz/npol
  //   );
  // }

  int out_ndat = ndat - 2*discard;

  // make sure we're not trying to access channels that don't exist
  if (idx > out_ndat || idy > nchan || idz > npol*npart) {
    return;
  }

  int in_offset;
  int out_offset;


  for (int ipart=idz/npol; ipart<npart; ipart+=npart_incr) {
    for (int ipol=idz%npol; ipol<npol; ipol+=npol_incr) {
      for (int ichan=idy; ichan < nchan; ichan += total_size_y) {
        for (int idat=idx; idat < out_ndat; idat += total_size_x) {
          in_offset = ipart*npol*nchan*ndat + ipol*nchan*ndat + ichan*ndat;
          out_offset = ipart*npol*nchan*out_ndat + ipol*nchan*out_ndat + ichan*out_ndat;
          if (resp == nullptr) {
            t_out[out_offset + idat] = t_in[in_offset + idat + discard];
          } else {
            t_out[out_offset + idat] = hipCmulf(resp[ichan*out_ndat + idat], t_in[in_offset + idat + discard]);
          }
        }
      }
    }
  }
}

/*!
 * fft shift an index. Returns -1 if ``idx`` is greater than ``ndat``
 * \method d_fft_shift_idx
 * \param idx the index to shift
 * \ndat the number of points about which to shift
 * \return circularly shifted index.
 */
__device__ int d_fft_shift_idx (int idx, int ndat)
{
  int ndat_2 = ndat / 2;
  if (idx >= ndat) {
    return -1;
  }
  if (idx >= ndat_2) {
    return idx - ndat_2;
  } else {
    return idx + ndat_2;
  }
}


/*!
 * Kernel for stitching together the result of forward FFTs, and multiplying
 * Response or ResponseProduct's internal buffer by stitched result.
 * \method k_response_stitch
 * \param f_in the frequency domain input data pointer.
 *    Dimensions are (npol*input_nchan, input_ndat).
 *    Here, input_ndat is equal to the size of the forward FFT.
 * \param response the Response or ResponseProduct's buffer
 * \param f_out the frequency domain output data pointer.
 *    Dimensions are (npol*1, output_ndat).
 *    Here, output_ndat is equal to the size of the backward FFT,
 *    which is in turn equal to input_nchan * input_ndat normalized by
 *    the oversampling factor.
 * \param os_discard the number of *complex samples* to discard
 *    from either side of the input spectra.
 * \param npol the number of polarisations
 * \param in_nchan the number of channels in the input data. The first dimension
 *    of the input data is in_nchan*npol.
 * \param in_ndat the second dimension of the input data.
 * \param out_ndat the second dimension of the output data.
 * \param pfb_dc_chan whether or not the DC channel of the PFB channeliser is
 *    present.
 * \param pfb_all_chan whether or not all the channels from the PFB channeliser
 *    are present.
 */
__global__ void k_response_stitch (
  float2* f_in,
  float2* response,
  float2* f_out,
  int os_discard,
  int npart,
  int npol,
  int in_nchan,
  int in_ndat,
  int out_ndat,
  bool pfb_dc_chan,
  bool pfb_all_chan
)
{
  int total_size_x = blockDim.x * gridDim.x; // for idat
  int total_size_y = blockDim.y * gridDim.y; // for ichan
  int total_size_z = blockDim.z * gridDim.z; // for ipol and ipart
  int npol_incr = total_size_z <= npol ? 1: npol;
  int npart_incr = total_size_z/npol == 0 ? 1: total_size_z/npol;


  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int idy = blockIdx.y*blockDim.y + threadIdx.y;
  int idz = blockIdx.z*blockDim.z + threadIdx.z;

  // don't overstep the data
  if (idx > in_ndat - os_discard || idy > in_nchan || idz > (npol*npart)) {
    return;
  }

  int in_ndat_keep = in_ndat - 2*os_discard;
  int in_ndat_keep_2 = in_ndat_keep / 2;

  int in_offset;
  int out_offset;

  int in_idx_bot;
  int in_idx_top;

  int out_idx_bot;
  int out_idx_top;


  for (int ipart=idz/npol; ipart < npart; ipart += npart_incr) {
    for (int ipol=idz%npol; ipol < npol; ipol += npol_incr) {
      for (int ichan=idy; ichan < in_nchan; ichan += total_size_y) {
        in_offset = ipart*npol*in_ndat*in_nchan + ipol*in_ndat*in_nchan + ichan*in_ndat;
        out_offset = ipart*npol*out_ndat + ipol*out_ndat;
        // std::cerr << "in_offset=" << in_offset << ", out_offset=" << out_offset << std::endl;

        for (int idat=idx; idat<in_ndat_keep_2; idat += total_size_x) {
          in_idx_top = idat;
          in_idx_bot = in_idx_top + (in_ndat - in_ndat_keep_2);

          out_idx_bot = idat + in_ndat_keep*ichan;
          out_idx_top = out_idx_bot + in_ndat_keep_2;

          if (pfb_dc_chan) {
            if (ichan == 0) {
              out_idx_top = idat;
              out_idx_bot = idat + (out_ndat - in_ndat_keep_2);
            } else {
              out_idx_bot = idat + in_ndat_keep*(ichan-1) + in_ndat_keep_2;
              out_idx_top = out_idx_bot + in_ndat_keep_2;
            }
          }

          // std::cerr << in_offset + in_idx_bot << ", " << in_offset + in_idx_top << std::endl;
          // std::cerr << out_offset + out_idx_bot << ", " << out_offset + out_idx_top << std::endl;
          //
          // if (in_offset + in_idx_bot > in_size ||
          //     out_offset + out_idx_top > out_size ||
          //     in_offset + in_idx_top > in_size ||
          //     out_offset + out_idx_bot > out_size) {
          //   std::cerr << "watch out!" << std::endl;
          // }
          // std::cerr << "in=[" << in_idx_bot << "," << in_idx_top << "] out=["
          //   << out_idx_bot << "," << out_idx_top << "]" << std::endl;

          f_out[out_offset + out_idx_bot] = hipCmulf(response[out_idx_bot], f_in[in_offset + in_idx_bot]);
          f_out[out_offset + out_idx_top] = hipCmulf(response[out_idx_top], f_in[in_offset + in_idx_top]);

          if (! pfb_all_chan && pfb_dc_chan && ichan == 0) {
            f_out[out_offset + out_idx_bot].x = 0.0;
            f_out[out_offset + out_idx_bot].y = 0.0;
          }
        }
      }
    }
  }
  // for (int ipol=idz; ipol < npol; ipol += total_size_z) {
  //   for (int ichan=idy; ichan < in_nchan; ichan += total_size_y) {
  //     for (int idat=idx; idat < (in_ndat - os_discard); idat += total_size_x) {
  //       if (idat < os_discard) {
  //         continue;
  //       }
  //       in_offset = ipol*in_nchan*in_ndat + ichan*in_ndat;
  //       out_offset = ipol*out_ndat;
  //
  //       in_idx_top = in_offset + (idat - os_discard);
  //       in_idx_bot = in_idx_top + (in_ndat - in_ndat_keep_2);
  //
  //       out_idx_bot = ichan*in_ndat_keep + (idat - os_discard);
  //       out_idx_top = out_idx_bot + in_ndat_keep_2;
  //
  //       if (pfb_dc_chan) {
  //         if (ichan == 0) {
  //           out_idx_bot = idat - os_discard;
  //           out_idx_top = out_idx_bot + out_ndat - in_ndat_keep_2;
  //         } else {
  //           out_idx_bot += in_ndat_keep_2;
  //           out_idx_top += in_ndat_keep_2;
  //         }
  //       }
  //       f_out[out_idx_bot + out_offset] = hipCmulf(response[out_idx_bot], f_in[in_idx_bot]);
  //       f_out[out_idx_top + out_offset] = hipCmulf(response[out_idx_top], f_in[in_idx_top]);
  //
  //       if (pfb_dc_chan && ! pfb_all_chan) {
  //         f_out[out_idx_top + out_offset].x = 0.0;
  //         f_out[out_idx_top + out_offset].y = 0.0;
  //       }
  //     }
  //   }
  // }
}


CUDA::InverseFilterbankEngineCUDA::InverseFilterbankEngineCUDA (hipStream_t _stream)
{
  stream = _stream;

  input_fft_length = 0;
  forward_fft_plan_setup = false;
  backward_fft_plan_setup = false;
  response = nullptr;
  fft_window = nullptr;

  pfb_dc_chan = 0;
  pfb_all_chan = 0;
  verbose = dsp::Observation::verbose;

  hipfftHandle plans[] = {forward, backward};
  int nplans = sizeof(plans) / sizeof(plans[0]);
  hipfftResult result;
  for (int i=0; i<nplans; i++) {
    result = hipfftCreate (&plans[i]);
    if (verbose) {
      std::cerr << "CUDA::InverseFilterbankEngineCUDA::InverseFilterbankEngineCUDA: i=" << i << " result=" << result << std::endl;
    }
    if (result != HIPFFT_SUCCESS) {
      throw CUFFTError (
        result,
        "CUDA::InverseFilterbankEngineCUDA::InverseFilterbankEngineCUDA",
        "hipfftCreate");
    }
  }
}

CUDA::InverseFilterbankEngineCUDA::~InverseFilterbankEngineCUDA ()
{
  hipfftHandle plans[] = {forward, backward};
  int nplans = sizeof(plans) / sizeof(plans[0]);

  hipfftResult result;
  for (int i=0; i<nplans; i++) {
    result = hipfftDestroy (plans[i]);
    if (verbose) {
      std::cerr << "CUDA::InverseFilterbankEngineCUDA::~InverseFilterbankEngineCUDA: i=" << i << " result=" << result << std::endl;
    }
    if (result == HIPFFT_INVALID_PLAN) {
      if (verbose) {
        std::cerr << "CUDA::InverseFilterbankEngineCUDA::~InverseFilterbankEngineCUDA: plan[" << i << "] was invalid" << std::endl;
      }
      // throw CUFFTError (
      //   result,
      //   "CUDA::InverseFilterbankEngineCUDA::InverseFilterbankEngineCUDA",
      //   "hipfftDestroy");
    }

  }
}

std::vector<hipfftResult> CUDA::InverseFilterbankEngineCUDA::setup_forward_fft_plan (
  unsigned _input_fft_length,
  unsigned _input_nchan,
  hipfftType _type_forward
)
{
  // setup forward batched plan
  int rank = 1; // 1D transform
  int n[] = {_input_fft_length}; /* 1d transforms of length 10 */
  int howmany = _input_nchan;
  int idist = _input_fft_length;
  int odist = _input_fft_length;
  int istride = 1;
  int ostride = 1;
  int *inembed = n, *onembed = n;
  hipfftResult result;
  std::vector<hipfftResult> results;

  // hipfftResult = hipfftPlanMany(hipfftHandle *plan, int rank, int *n, int *inembed,
  //     int istride, int idist, int *onembed, int ostride,
  //     int odist, hipfftType type, int batch)

  // result = hipfftPlan1d (&forward, input_fft_length, type_forward, 1);
  result = hipfftPlanMany(
    &forward, rank, n,
    inembed, istride, idist,
    onembed, ostride, odist,
    _type_forward, howmany);
  results.push_back(result);

  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::InverseFilterbankEngineCUDA::setup_forward_fft_plan",
                      "hipfftPlanMany(forward)");

  result = hipfftSetStream (forward, stream);
  results.push_back(result);

  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::InverseFilterbankEngineCUDA::setup_forward_fft_plan",
          "hipfftSetStream(forward)");
  forward_fft_plan_setup = true;
  return results;
}

std::vector<hipfftResult> CUDA::InverseFilterbankEngineCUDA::setup_backward_fft_plan (
  unsigned _output_fft_length,
  unsigned _output_nchan
)
{
  // setup forward batched plan
  int rank = 1; // 1D transform
  int n[] = { _output_fft_length}; /* 1d transforms of length 10 */
  int howmany = _output_nchan;
  int idist = _output_fft_length;
  int odist = _output_fft_length;
  int istride = 1;
  int ostride = 1;
  int *inembed = n, *onembed = n;
  hipfftResult result;
  std::vector<hipfftResult> results;


  // result = hipfftPlan1d (&backward, output_fft_length, HIPFFT_C2C, 1);
  result = hipfftPlanMany(
    &backward, rank, n,
    inembed, istride, idist,
    onembed, ostride, odist,
    HIPFFT_C2C, howmany);

  results.push_back(result);

  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::InverseFilterbankEngineCUDA::setup_backward_fft_plan",
                      "hipfftPlan1d(backward)");

  result = hipfftSetStream (backward, stream);
  results.push_back(result);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::InverseFilterbankEngineCUDA::setup_backward_fft_plan",
                      "hipfftSetStream(backward)");
  backward_fft_plan_setup = true;

  return results;
}


void CUDA::InverseFilterbankEngineCUDA::setup (
  const dsp::TimeSeries* input,
  dsp::TimeSeries* output,
  const Rational& os_factor,
  unsigned _input_fft_length,
  unsigned _output_fft_length,
  unsigned _input_discard_pos,
  unsigned _input_discard_neg,
  unsigned _output_discard_pos,
  unsigned _output_discard_neg,
  bool _pfb_dc_chan,
  bool _pfb_all_chan
)
{
  type_forward = HIPFFT_C2C;
  n_per_sample = 1;

  if (input->get_state() == Signal::Nyquist) {
    type_forward = HIPFFT_R2C;
    n_per_sample = 2;
  }

  pfb_dc_chan = _pfb_dc_chan;
  pfb_all_chan = _pfb_all_chan;

  input_npol = input->get_npol();
  input_nchan = input->get_nchan();
  output_nchan = output->get_nchan();

  input_fft_length = _input_fft_length;
  output_fft_length = _output_fft_length;

  input_discard_pos = _input_discard_pos;
  input_discard_neg = _input_discard_neg;
  output_discard_pos = _output_discard_pos;
  output_discard_neg = _output_discard_neg;

  input_discard_total = n_per_sample*(input_discard_neg + input_discard_pos);
  input_sample_step = input_fft_length - input_discard_total;

  output_discard_total = n_per_sample*(output_discard_neg + output_discard_pos);
  output_sample_step = output_fft_length - output_discard_total;

  input_os_keep = os_factor.normalize(input_fft_length);
  input_os_discard = input_fft_length - input_os_keep;

  setup_forward_fft_plan(
    input_fft_length, input_nchan, type_forward
  );

  setup_backward_fft_plan(
    output_fft_length, output_nchan
  );
}



void CUDA::InverseFilterbankEngineCUDA::setup (dsp::InverseFilterbank* filterbank)
{

  const dsp::TimeSeries* input = filterbank->get_input();
  dsp::TimeSeries* output = filterbank->get_output();

  setup (
    input,
    output,
    filterbank->get_oversampling_factor(),
    filterbank->get_input_fft_length(),
    filterbank->get_output_fft_length(),
    filterbank->get_input_discard_pos(),
    filterbank->get_input_discard_neg(),
    filterbank->get_output_discard_pos(),
    filterbank->get_output_discard_neg(),
    filterbank->get_pfb_dc_chan(),
    filterbank->get_pfb_all_chan()
  );
}


void CUDA::InverseFilterbankEngineCUDA::set_scratch (float* )
{ }


void CUDA::InverseFilterbankEngineCUDA::perform (
  const dsp::TimeSeries* in,
  dsp::TimeSeries* out,
  uint64_t npart
  // uint64_t in_step,
  // uint64_t out_step
)
{
  // in_step is input_sample_step
  // out_step is output_sample_step


  // typedef CUDA::cufftTypeMap<type_forward> type_map;
  //
  // // in_step and out_step are unused, as they get calculated in setup
  // dim3 grid (1, nchan, npol*npart);
  // dim3 threads (1024, 1, 1);
  // grid.x = (ndat / threads.x) + 1;
  //
  // typename type_map::input_type* in_device;
  // hipfftComplex* out_device;
  //
  // k_overlap_discard<<<grid, threads, 0, stream>>>(
  //   in_device, apod_device, in_scratch_device,
  //   input_discard_total, npart, input_npol, input_nchan, input_fft_length
  // );
  //
  // for (uint64_t ipart=0; ipart<npart; ipart++)
  // {
  //   type_map::cufftExec(
  //     forward,
  //     in_scratch_device + ipart * input_fft_length * input_nchan,
  //     in_scratch_device + ipart * input_fft_length * input_nchan,
  //     HIPFFT_FORWARD);
  // }
  //
  // k_response_stitch<<<grid, threads, 0, stream>>>(
  //   in_device, resp_device, out_device, os_discard, npart,
  //   npol, nchan, in_ndat, out_ndat, pfb_dc_chan, pfb_all_chan);
  // );
  //
  // for (uint64_t ipart=0; ipart<npart; ipart++)
  // {
  //   hipfftExecC2C(
  //     backward,
  //     in_scratch_device + ipart * output_fft_length * output_nchan,
  //     in_scratch_device + ipart * output_fft_length * output_nchan,
  //     HIPFFT_BACKWARD);
  // }
  //
  // k_overlap_discard<<<grid, threads, 0, stream>>>(
  //   in_scratch, nullptr, out_device,
  //   output_discard_total, npart, input_npol, output_nchan, output_fft_length
  // );

}

void CUDA::InverseFilterbankEngineCUDA::finish ()
{
  if (verbose) {
    std::cerr << "dsp::InverseFilterbankEngineCPU::finish" << std::endl;
  }
}


//! This method is static
void CUDA::InverseFilterbankEngineCUDA::apply_k_response_stitch (
  std::vector< std::complex<float> >& in,
  std::vector< std::complex<float> >& response,
  std::vector< std::complex<float> >& out,
  Rational os_factor,
  unsigned npart,
  unsigned npol,
  unsigned nchan,
  unsigned ndat,
  bool pfb_dc_chan,
  bool pfb_all_chan)
{
  float2* in_device;
  float2* resp_device;
  float2* out_device;

  unsigned in_ndat = ndat;
  unsigned os_keep = os_factor.normalize(in_ndat);
  unsigned os_discard = (in_ndat - os_keep)/2;
  unsigned out_ndat = nchan * os_keep;
  unsigned out_size = npart * npol * out_ndat;
  unsigned in_size = npart * npol * nchan * ndat;

  if (out.size() != out_size) {
    out.resize(out_size);
  }

  size_t sz = sizeof(float2);

  hipMalloc((void **) &in_device, in_size*sz);
  hipMalloc((void **) &resp_device, out_ndat*sz);
  hipMalloc((void **) &out_device, out_size*sz);

  hipMemcpy(
    in_device, (float2*) in.data(), in_size*sz, hipMemcpyHostToDevice);
  hipMemcpy(
    resp_device, (float2*) response.data(), out_ndat*sz, hipMemcpyHostToDevice);

  // 10 is sort of arbitrary here.
  dim3 grid (1, nchan, npart*npol);
  dim3 threads (in_ndat, 1, 1);

  k_response_stitch<<<grid, threads>>>(
    in_device, resp_device, out_device, os_discard, npart,
    npol, nchan, in_ndat, out_ndat, pfb_dc_chan, pfb_all_chan);

  check_error( "CUDA::InverseFilterbankEngineCUDA::apply_k_response_stitch" );

  hipMemcpy((float2*) out.data(), out_device, out_size*sz, hipMemcpyDeviceToHost);

  hipFree(in_device);
  hipFree(resp_device);
  hipFree(out_device);

}

//! This method is static
void CUDA::InverseFilterbankEngineCUDA::apply_k_apodization_overlap (
  std::vector< std::complex<float> >& in,
  std::vector< std::complex<float> >& apodization,
  std::vector< std::complex<float> >& out,
  unsigned discard,
  unsigned npart,
  unsigned npol,
  unsigned nchan,
  unsigned ndat
)
{
  float2* in_device;
  float2* apod_device;
  float2* out_device;

  size_t sz = sizeof(float2);

  unsigned out_ndat = ndat - 2*discard;
  unsigned in_size = npart * npol * nchan * ndat;
  unsigned out_size = npart * npol * nchan * out_ndat;
  unsigned apod_size = nchan * out_ndat;

  hipMalloc((void **) &in_device, in_size*sz);
  hipMalloc((void **) &apod_device, apod_size*sz);
  hipMalloc((void **) &out_device, out_size*sz);

  hipMemcpy(
    in_device, (float2*) in.data(), in_size*sz, hipMemcpyHostToDevice);
  hipMemcpy(
    apod_device, (float2*) apodization.data(), apod_size*sz, hipMemcpyHostToDevice);

  dim3 grid (1, nchan, npol*npart);
  dim3 threads (1024, 1, 1);
  grid.x = (ndat / threads.x) + 1;


  k_overlap_discard<<<grid, threads>>>(
    in_device, apod_device, out_device, discard, npart, npol, nchan, ndat);
  check_error( "CUDA::InverseFilterbankEngineCUDA::apply_k_apodization_overlap" );

  hipMemcpy((float2*) out.data(), out_device, out_size*sz, hipMemcpyDeviceToHost);

  hipFree(in_device);
  hipFree(apod_device);
  hipFree(out_device);

}


void CUDA::InverseFilterbankEngineCUDA::apply_k_overlap_discard (
  std::vector< std::complex<float> >& in,
  std::vector< std::complex<float> >& out,
  unsigned discard,
  unsigned npart,
  unsigned npol,
  unsigned nchan,
  unsigned ndat
)
{
  float2* in_device;
  float2* out_device;

  size_t sz = sizeof(float2);

  unsigned out_ndat = ndat - 2*discard;

  unsigned in_size = npart * npol * nchan * ndat;
  unsigned out_size = npart * npol * nchan * out_ndat;

  hipMalloc((void **) &in_device, in_size*sz);
  hipMalloc((void **) &out_device, out_size*sz);

  hipMemcpy(
    in_device, (float2*) in.data(), in_size*sz, hipMemcpyHostToDevice);

  dim3 grid (1, nchan, npol*npart);
  dim3 threads (1024, 1, 1);

  grid.x = (ndat / threads.x) + 1;

  // std::cerr << grid.x << " " << grid.y << " " << grid.z << std::endl;
  // std::cerr << threads.x << " " << threads.y << " " << threads.z << std::endl;

  k_overlap_discard<<<grid, threads>>>(
    in_device, nullptr, out_device, discard, npart, npol, nchan, ndat);

  check_error( "CUDA::InverseFilterbankEngineCUDA::apply_k_overlap_discard" );

  hipMemcpy((float2*) out.data(), out_device, out_size*sz, hipMemcpyDeviceToHost);

  hipFree(in_device);
  hipFree(out_device);

}


void CUDA::InverseFilterbankEngineCUDA::apply_cufft_backward (
  std::vector< std::complex<float> >& in,
  std::vector< std::complex<float> >& out
)
{
  if (! backward_fft_plan_setup) {
    throw "CUDA::InverseFilterbankEngineCUDA::apply_cufft_backward: Backward FFT plan not setup";
  }

  hipfftComplex* in_cufft;
  hipfftComplex* out_cufft;

  size_t sz = sizeof(hipfftComplex);

  hipMalloc((void **) &in_cufft, in.size()*sz);
  hipMalloc((void **) &out_cufft, out.size()*sz);

  hipMemcpy(
    in_cufft, (hipfftComplex*) in.data(), in.size()*sz, hipMemcpyHostToDevice);

  hipfftExecC2C(backward, in_cufft, out_cufft, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  check_error( "CUDA::InverseFilterbankEngineCUDA::apply_k_overlap_discard" );

  hipMemcpy(
    (hipfftComplex*) out.data(), out_cufft, out.size()*sz, hipMemcpyDeviceToHost);

  hipFree(in_cufft);
  hipFree(out_cufft);

}
